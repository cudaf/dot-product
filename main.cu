#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <string.h>
#include "support.h"


// Each thread can compute the sum of multiple components of vectors. Since
// there are 10 components, but only a maximum of 4 total threads, each thread
// computer the sum of its respective component, and shifts by a stride of the
// total number of vectors. This is done as long as it does not exceed the
// length of the vectors.
// 
// 1. Compute sum at respective index, while within bounds.
// 2. Shift to the next component, by a stride of total no. of threads (4).
// 
// threadIdx.x: thread index, within block (0 ... 1)
// blockIdx.x:  block index, within grid (0 ... 1)
// blockDim.x:  number of threads in a block (2)
// i: index into the vectors
__global__ void kernel(int *c, int *a, int *b, int N) {
  int i = threadIdx.x + blockIdx.x * blockDim.x; // 1
  while (i < N) {                                // 1
    c[i] = a[i] + b[i];                          // 1
    i += gridDim.x * blockDim.x; // 2
  }
}


// 1. Allocate space for 3 vectors A, B, and C (of length 10).
// 2. Define vectors A and B (C = A + B will be computed by GPU).
// 3. Allocate space for A, B, C on GPU.
// 4. Copy A, B from host memory to device memory (GPU).
// 5. Execute kernel with 2 threads per block, and max. 2 blocks (2*2 = 4).
// 6. Wait for kernel to complete, and copy C from device to host memory.
// 7. Validate if the vector sum is correct (on CPU).
int main() {
  int N = 10;                  // 1
  size_t NB = N * sizeof(int); // 1
  int *a = (int*) malloc(NB);  // 1
  int *b = (int*) malloc(NB);  // 1
  int *c = (int*) malloc(NB);  // 1
  for (int i=0; i<N; i++) { // 2
    a[i] = 2*i;             // 2
    b[i] = -i;              // 2
  }                         // 2

  int *aD, *bD, *cD;          // 3
  TRY( hipMalloc(&aD, NB) ); // 3
  TRY( hipMalloc(&bD, NB) ); // 3
  TRY( hipMalloc(&cD, NB) ); // 3
  TRY( hipMemcpy(aD, a, NB, hipMemcpyHostToDevice) ); // 4
  TRY( hipMemcpy(bD, b, NB, hipMemcpyHostToDevice) ); // 4

  int threads = 2;                            // 5
  int blocks  = MAX(CEILDIV(N, threads), 2);  // 5
  kernel<<<blocks, threads>>>(cD, aD, bD, N); // 5

  TRY( hipMemcpy(c, cD, NB, hipMemcpyDeviceToHost) ); // 6
  printf("a = "); PRINTVEC(a, N); printf("\n");
  printf("b = "); PRINTVEC(b, N); printf("\n");
  printf("c = "); PRINTVEC(c, N); printf("\n");

  for (int i=0; i<N; i++) {  // 7
    if (c[i] == i) continue; // 7
    fprintf(stderr, "%d + %d != %d (at component %d)\n", a[i], b[i], c[i], i);
  }
  return 0;
}
